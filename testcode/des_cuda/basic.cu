#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <sys/time.h>
#include <time.h>

#define SCALE 49000
#define BLOCK_LENGTH 512
#define DIM 512
#define MASK(bit) (~(-1L<<bit))
#define HIHALF(num,bit) num>>(bit/2)
#define LOHALF(num,bit) num&MASK(bit/2)
#define HIGH(num) HIHALF(num,sizeof(num)*8)
#define LOW(num) LOHALF(num,sizeof(num)*8)
#define LONGCAT(hi,low,bit) (((long long)hi<<(bit/2))+(unsigned)low)
#define LCROL(num,bit) ((num&MASK(bit-1))<<1)+((num&(1<<bit-1))!=0)

__constant__ long long CUIP_Table[64] =
{
    1L<<64-58, 1L<<64-50, 1L<<64-42, 1L<<64-34, 1L<<64-26, 1L<<64-18, 1L<<64-10, 1L<<64-2,
    1L<<64-60, 1L<<64-52, 1L<<64-44, 1L<<64-36, 1L<<64-28, 1L<<64-20, 1L<<64-12, 1L<<64-4,
    1L<<64-62, 1L<<64-54, 1L<<64-46, 1L<<64-38, 1L<<64-30, 1L<<64-22, 1L<<64-14, 1L<<64-6,
    1L<<64-64, 1L<<64-56, 1L<<64-48, 1L<<64-40, 1L<<64-32, 1L<<64-24, 1L<<64-16, 1L<<64-8,
    1L<<64-57, 1L<<64-49, 1L<<64-41, 1L<<64-33, 1L<<64-25, 1L<<64-17,  1L<<64-9, 1L<<64-1,
    1L<<64-59, 1L<<64-51, 1L<<64-43, 1L<<64-35, 1L<<64-27, 1L<<64-19, 1L<<64-11, 1L<<64-3,
    1L<<64-61, 1L<<64-53, 1L<<64-45, 1L<<64-37, 1L<<64-29, 1L<<64-21, 1L<<64-13, 1L<<64-5,
    1L<<64-63, 1L<<64-55, 1L<<64-47, 1L<<64-39, 1L<<64-31, 1L<<64-23, 1L<<64-15, 1L<<64-7
};

//逆初始置换表IP^-1
__constant__ long long CUIP_1_Table[64] =
{
    1L<<64-40, 1L<<64-8, 1L<<64-48, 1L<<64-16, 1L<<64-56, 1L<<64-24, 1L<<64-64, 1L<<64-32,
    1L<<64-39, 1L<<64-7, 1L<<64-47, 1L<<64-15, 1L<<64-55, 1L<<64-23, 1L<<64-63, 1L<<64-31,
    1L<<64-38, 1L<<64-6, 1L<<64-46, 1L<<64-14, 1L<<64-54, 1L<<64-22, 1L<<64-62, 1L<<64-30,
    1L<<64-37, 1L<<64-5, 1L<<64-45, 1L<<64-13, 1L<<64-53, 1L<<64-21, 1L<<64-61, 1L<<64-29,
    1L<<64-36, 1L<<64-4, 1L<<64-44, 1L<<64-12, 1L<<64-52, 1L<<64-20, 1L<<64-60, 1L<<64-28,
    1L<<64-35, 1L<<64-3, 1L<<64-43, 1L<<64-11, 1L<<64-51, 1L<<64-19, 1L<<64-59, 1L<<64-27,
    1L<<64-34, 1L<<64-2, 1L<<64-42, 1L<<64-10, 1L<<64-50, 1L<<64-18, 1L<<64-58, 1L<<64-26,
    1L<<64-33, 1L<<64-1, 1L<<64-41,   1L<<64-9, 1L<<64-49, 1L<<64-17, 1L<<64-57, 1L<<64-25
};

//扩充置换表E
__constant__ long long CUE_Table[48] =
{
    1L<<32-32,  1L<<32-1,   1L<<32-2,   1L<<32-3,   1L<<32-4,   1L<<32-5,
    1L<<32-4,   1L<<32-5,   1L<<32-6,   1L<<32-7,   1L<<32-8,   1L<<32-9,
    1L<<32-8,   1L<<32-9, 1L<<32-10, 1L<<32-11, 1L<<32-12, 1L<<32-13,
    1L<<32-12, 1L<<32-13, 1L<<32-14, 1L<<32-15, 1L<<32-16, 1L<<32-17,
    1L<<32-16, 1L<<32-17, 1L<<32-18, 1L<<32-19, 1L<<32-20, 1L<<32-21,
    1L<<32-20, 1L<<32-21, 1L<<32-22, 1L<<32-23, 1L<<32-24, 1L<<32-25,
    1L<<32-24, 1L<<32-25, 1L<<32-26, 1L<<32-27, 1L<<32-28, 1L<<32-29,
    1L<<32-28, 1L<<32-29, 1L<<32-30, 1L<<32-31, 1L<<32-32,   1L<<32-1
};

//置换函数P
__constant__ unsigned int CUP[32] =
{
    1L<<32-16,   1L<<32-7, 1L<<32-20, 1L<<32-21,
    1L<<32-29, 1L<<32-12, 1L<<32-28, 1L<<32-17,
    1L<<32-1, 1L<<32-15, 1L<<32-23, 1L<<32-26,
    1L<<32-5, 1L<<32-18, 1L<<32-31, 1L<<32-10,
    1L<<32-2,   1L<<32-8, 1L<<32-24, 1L<<32-14,
    1L<<32-32, 1L<<32-27,   1L<<32-3,   1L<<32-9,
    1L<<32-19, 1L<<32-13, 1L<<32-30,   1L<<32-6,
    1L<<32-22, 1L<<32-11,   1L<<32-4, 1L<<32-25
};

//S盒
__constant__ const char CUS[8][4][16] = 
{
    // S1
    14,  4, 13,  1,  2, 15, 11,  8,  3, 10,  6, 12,  5,  9,  0,  7,
    0, 15,  7,  4, 14,  2, 13,  1, 10,  6, 12, 11,  9,  5,  3,  8,
    4,  1, 14,  8, 13,  6,  2, 11, 15, 12,  9,  7,  3, 10,  5,  0,
    15, 12,  8,  2,  4,  9,  1,  7,  5, 11,  3, 14, 10,  0,  6, 13,
    // S2 
    15,  1,  8, 14,  6, 11,  3,  4,  9,  7,  2, 13, 12,  0,  5, 10,
    3, 13,  4,  7, 15,  2,  8, 14, 12,  0,  1, 10,  6,  9, 11,  5,
    0, 14,  7, 11, 10,  4, 13,  1,  5,  8, 12,  6,  9,  3,  2, 15,
    13,  8, 10,  1,  3, 15,  4,  2, 11,  6,  7, 12,  0,  5, 14,  9,
    // S3 
    10,  0,  9, 14,  6,  3, 15,  5,  1, 13, 12,  7, 11,  4,  2,  8,
    13,  7,  0,  9,  3,  4,  6, 10,  2,  8,  5, 14, 12, 11, 15,  1,
    13,  6,  4,  9,  8, 15,  3,  0, 11,  1,  2, 12,  5, 10, 14,  7,
    1, 10, 13,  0,  6,  9,  8,  7,  4, 15, 14,  3, 11,  5,  2, 12,
    // S4 
    7, 13, 14,  3,  0,  6,  9, 10,  1,  2,  8,  5, 11, 12,  4, 15,
    13,  8, 11,  5,  6, 15,  0,  3,  4,  7,  2, 12,  1, 10, 14,  9,
    10,  6,  9,  0, 12, 11,  7, 13, 15,  1,  3, 14,  5,  2,  8,  4,
    3, 15,  0,  6, 10,  1, 13,  8,  9,  4,  5, 11, 12,  7,  2, 14,
    // S5 
    2, 12,  4,  1,  7, 10, 11,  6,  8,  5,  3, 15, 13,  0, 14,  9,
    14, 11,  2, 12,  4,  7, 13,  1,  5,  0, 15, 10,  3,  9,  8,  6,
    4,  2,  1, 11, 10, 13,  7,  8, 15,  9, 12,  5,  6,  3,  0, 14,
    11,  8, 12,  7,  1, 14,  2, 13,  6, 15,  0,  9, 10,  4,  5,  3,
    // S6 
    12,  1, 10, 15,  9,  2,  6,  8,  0, 13,  3,  4, 14,  7,  5, 11,
    10, 15,  4,  2,  7, 12,  9,  5,  6,  1, 13, 14,  0, 11,  3,  8,
    9, 14, 15,  5,  2,  8, 12,  3,  7,  0,  4, 10,  1, 13, 11,  6,
    4,  3,  2, 12,  9,  5, 15, 10, 11, 14,  1,  7,  6,  0,  8, 13,
    // S7 
    4, 11,  2, 14, 15,  0,  8, 13,  3, 12,  9,  7,  5, 10,  6,  1,
    13,  0, 11,  7,  4,  9,  1, 10, 14,  3,  5, 12,  2, 15,  8,  6,
    1,  4, 11, 13, 12,  3,  7, 14, 10, 15,  6,  8,  0,  5,  9,  2,
    6, 11, 13,  8,  1,  4, 10,  7,  9,  5,  0, 15, 14,  2,  3, 12,
    // S8 
    13,  2,  8,  4,  6, 15, 11,  1, 10,  9,  3, 14,  5,  0, 12,  7,
    1, 15, 13,  8, 10,  3,  7,  4, 12,  5,  6, 11,  0, 14,  9,  2,
    7, 11,  4,  1,  9, 12, 14,  2,  0,  6, 10, 13, 15,  3,  5,  8,
    2,  1, 14,  7,  4, 10,  8, 13, 15, 12,  9,  0,  3,  5,  6, 11
};

//置换选择1
__constant__ long long PC_1[56] =
{
    1L<<64-57,1L<<64-49,1L<<64-41,1L<<64-33,1L<<64-25,1L<<64-17,1L<<64-9,
    1L<<64-1,1L<<64-58,1L<<64-50,1L<<64-42,1L<<64-34,1L<<64-26,1L<<64-18,
    1L<<64-10,1L<<64-2,1L<<64-59,1L<<64-51,1L<<64-43,1L<<64-35,1L<<64-27,
    1L<<64-19,1L<<64-11,1L<<64-3,1L<<64-60,1L<<64-52,1L<<64-44,1L<<64-36,
    1L<<64-63,1L<<64-55,1L<<64-47,1L<<64-39,1L<<64-31,1L<<64-23,1L<<64-15,
    1L<<64-7,1L<<64-62,1L<<64-54,1L<<64-46,1L<<64-38,1L<<64-30,1L<<64-22,
    1L<<64-14,1L<<64-6,1L<<64-61,1L<<64-53,1L<<64-45,1L<<64-37,1L<<64-29,
    1L<<64-21,1L<<64-13,1L<<64-5,1L<<64-28,1L<<64-20,1L<<64-12,1L<<64-4
};

//置换选择2
__constant__ long long PC_2[48] =
{
    1L<<56-14,1L<<56-17,1L<<56-11,1L<<56-24,1L<<56-1,1L<<56-5,
    1L<<56-3,1L<<56-28,1L<<56-15,1L<<56-6,1L<<56-21,1L<<56-10,
    1L<<56-23,1L<<56-19,1L<<56-12,1L<<56-4,1L<<56-26,1L<<56-8,
    1L<<56-16,1L<<56-7,1L<<56-27,1L<<56-20,1L<<56-13,1L<<56-2,
    1L<<56-41,1L<<56-52,1L<<56-31,1L<<56-37,1L<<56-47,1L<<56-55,
    1L<<56-30,1L<<56-40,1L<<56-51,1L<<56-45,1L<<56-33,1L<<56-48,
    1L<<56-44,1L<<56-49,1L<<56-39,1L<<56-56,1L<<56-34,1L<<56-53,
    1L<<56-46,1L<<56-42,1L<<56-50,1L<<56-36,1L<<56-29,1L<<56-32
};
__constant__ int MOVE_TIMES[16] = {1,1,2,2,2,2,2,2,1,2,2,2,2,2,2,1};

__device__ float getnextrand(hiprandState *state)
{
  return (float)(hiprand_uniform(state));
}

__device__ long long getnextrandscaled(hiprandState *state, int scale)
{
  return (long) scale * getnextrand(state)*scale;
}

__global__ void initCurand(hiprandState *state)
{
    int idx = threadIdx.x;
    hiprand_init(idx, 0, 0, &state[idx]);
}


__device__ long long des_cuda_applypc(long long key,long long*pc,int len)
{
    long long ret=0;
    ret=(key&pc[0])!=0?1:0;
    for(int i=1; i<len; i++)
    {
        ret<<=1;
        ret+=(key&pc[i])!=0?1:0;
    }
    return ret;
}

__device__ int des_cuda_applyPBox(int key)
{
    int ret=0;
    unsigned int *pc = CUP;
    int i=0;
    ret=(key&pc[0])!=0?1:0;
    for(i=1; i<32; i++)
    {
        ret<<=1;
        ret+=(key&pc[i])!=0?1:0;
    }
    return ret;
}

__device__ long long des_lrol(int *hi,int *low,int loop)
{
    while(loop>0)
    {
        *hi = LCROL(*hi,28);
        *low = LCROL(*low,28);
        loop--;
    }
    long long ret = LONGCAT(*hi,*low,56);
    return ret;
}

__device__ int des_ckey(long long key,long long *store)//生成轮换key
{
    long long tmp = des_cuda_applypc(key,PC_1,56);
    int hi = HIHALF(tmp,56);
    int low = LOHALF(tmp,56);
    for(int i=0; i<16; i++)
    {
        tmp = des_lrol(&hi,&low,MOVE_TIMES[i]);
        store[i]=des_cuda_applypc(tmp,PC_2,48);
    }
    return 0;
}

__device__ int des_cuda_applySBox(long long data)
{
    int tmp,part,i,row,col;
    tmp=0;
    for(i=0; i<8; i++)
    {
        row = ((data&0x20)>>4)+(data&0x1);
        col = (data&0x1E)>>1;
        data>>=6;
        part=CUS[7-i][row][col];
        part<<=(4*i);
        tmp+=part;
    }
    return tmp;
}

__device__ long long des_cuda_one(long long data,long long key)
{
    int hi=HIGH(data);
    int low=LOW(data);
    long long tmp;int sub;
    tmp = des_cuda_applypc(low,CUE_Table,48);
    tmp^=key;
    sub = des_cuda_applySBox(tmp);
    sub = des_cuda_applyPBox(sub);
    sub ^=hi;tmp = LONGCAT(low,sub,64);
    return tmp;
}

__global__ void des_cuda(hiprandState*state,long long *data)
{
    //int idx = threadIdx.x;
    int idx=blockDim.x * blockIdx.x + threadIdx.x;
    
    if(idx>=BLOCK_LENGTH*DIM) return; 
    long long t, ckey[16];int tim=0;
    while(tim<=(1<<7))
    {
        long long key= getnextrandscaled(&state[idx], SCALE);
        des_ckey(key,ckey);t = des_cuda_applypc(data[idx],CUIP_Table,64);
        for(int i=0;i<16;i++) t = des_cuda_one(t,ckey[i]);
        int h=HIGH(t),l=LOW(t);t = LONGCAT(l,h,64);
        data[idx] = des_cuda_applypc(t,CUIP_1_Table,64);
        tim++;
    }
}

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

typedef long long int64;

int main() 
{
    hiprandState *devState;
    hipMalloc((void**)&devState, DIM*BLOCK_LENGTH * sizeof(hiprandState));
    cudaCheckErrors("cudamalloc1");
    initCurand<<<DIM,BLOCK_LENGTH>>>(devState); 
    hipDeviceSynchronize();
    cudaCheckErrors("cudamalloc2");

    int64*data=new int64[BLOCK_LENGTH*DIM];int size=DIM*BLOCK_LENGTH*sizeof(int64);
    for(int i=0;i<BLOCK_LENGTH*DIM;i++) data[i]=rand();
    int64*d_data;hipMalloc((void**)&d_data,size);
    hipMemcpy(d_data,data,size,hipMemcpyHostToDevice);
    cudaCheckErrors("cudamalloc3");
    struct timeval tstart, tend;
    gettimeofday(&tstart, NULL);    
    des_cuda<<<DIM,BLOCK_LENGTH>>>(devState,d_data);
    hipDeviceSynchronize();
    cudaCheckErrors("cudamalloc4");

    hipMemcpy(data,d_data,size,hipMemcpyDeviceToHost);
    gettimeofday(&tend, NULL);
    long long uses = 1000000 * (tend.tv_sec - tstart.tv_sec) + (tend.tv_usec - tstart.tv_usec);
    printf("loop time: %lld\n", uses);  
    cudaCheckErrors("cudamalloc5");
    for(int i=0;i<10;i++) printf("i: %d value: %lld\n",i,data[i]);
}
