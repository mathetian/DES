#include "hip/hip_runtime.h"
#include "DESCuda.h"

#include <openssl/rand.h>
#include <openssl/des.h>

#include <iostream>
#include <sstream>
using namespace std;

#include <TimeStamp.h>

__device__ int GenerateKey(uint64_t key, uint64_t * store)
{
    uint32_t c, d, t, s, t2;
    uint64_t tmp;
    /**c: low 32 bits, d high 32 bits**/
    c = ((1ull << 32) - 1) & key;
    d = (key >> 32);

    PERM_OP (d,c,t,4,0x0f0f0f0fL);
    HPERM_OP(c,t, -2,0xcccc0000L);
    HPERM_OP(d,t, -2,0xcccc0000L);
    PERM_OP (d,c,t,1,0x55555555L);
    PERM_OP (c,d,t,8,0x00ff00ffL);
    PERM_OP (d,c,t,1,0x55555555L);

    d =	(((d&0x000000ffL)<<16L)| (d&0x0000ff00L)     |
         ((d&0x00ff0000L)>>16L)|((c&0xf0000000L)>>4L));
    c&=0x0fffffffL;

    //one round, 0.25s*16=4s

    RoundKey0(0);
    RoundKey0(1) ;
    RoundKey1(2) ;
    RoundKey1(3);
    RoundKey1(4);
    RoundKey1(5) ;
    RoundKey1(6) ;
    RoundKey1(7);
    RoundKey0(8);
    RoundKey1(9) ;
    RoundKey1(10);
    RoundKey1(11);
    RoundKey1(12);
    RoundKey1(13);
    RoundKey1(14);
    RoundKey0(15);

    return 0;
}

__global__ void Gee(uint64_t * store)
{
    //14969965219234971648 0xcfc0000d78740000L
    //14897907633854087168 0xcec0000f7c740000L
    //uint64_t key=0x0E0E0E0E0E0E0E02;
    uint64_t key=0x020E0E0E0E0E0E0E;
    GenerateKey(key,store);
}

__device__ uint64_t DESOneTime(uint64_t * roundKeys)
{
    uint64_t rs;
    uint32_t right = plRight, left = plLeft;

    IP(right, left);

    left  = ROTATE(left,29)&0xffffffffL;
    right = ROTATE(right,29)&0xffffffffL;

    D_ENCRYPT(left,right, 0);
    D_ENCRYPT(right,left, 1);
    D_ENCRYPT(left,right, 2);
    D_ENCRYPT(right,left, 3);
    D_ENCRYPT(left,right, 4);
    D_ENCRYPT(right,left, 5);
    D_ENCRYPT(left,right, 6);
    D_ENCRYPT(right,left, 7);
    D_ENCRYPT(left,right, 8);
    D_ENCRYPT(right,left, 9);
    D_ENCRYPT(left,right,10);
    D_ENCRYPT(right,left,11);
    D_ENCRYPT(left,right,12);
    D_ENCRYPT(right,left,13);
    D_ENCRYPT(left,right,14);
    D_ENCRYPT(right,left,15);

    left  = ROTATE(left,3)&0xffffffffL;
    right = ROTATE(right,3)&0xffffffffL;

    FP(right, left);

    rs=(((uint64_t)right)<<32)|left; //why, who can explain it
    return rs;
}

/**
	DESEncrypt was used to conduct basic experiment
**/
__global__ void DESEncrypt(uint64_t *data)
{
    /**Don't know why should use it.**/
    ((uint64_t *)des_SP)[threadIdx.x] = ((uint64_t *)des_d_sp_c)[threadIdx.x];
#if MAX_THREAD == 128
    ((uint64_t *)des_SP)[threadIdx.x+128] = ((uint64_t *)des_d_sp_c)[threadIdx.x+128];
#endif

    __syncthreads();

    register uint64_t key = data[TX];
    uint64_t roundKeys[16];

    for(int i = 0; i < (1<<8); i++)
    {
        GenerateKey(key,roundKeys);
        key = DESOneTime(roundKeys);
    }

    data[TX]=key;

    __syncthreads();
}

/**
	DESGeneratorCUDA, the really entrance function
**/
__global__ void  DESGeneratorCUDA(uint64_t * data)
{
    for(int i=0; i<256; i++)
    {
        ((uint64_t *)des_SP)[i] = ((uint64_t *)des_d_sp_c)[i];
    }

    /*((uint64_t *)des_SP)[threadIdx.x] = ((uint64_t *)des_d_sp_c)[threadIdx.x];*/

    __syncthreads();

    register uint64_t m_nIndex = data[TX];
    uint64_t roundKeys[16];

    /**
    	Sorry, I didn't find how to change the device
    	value in general CODE, so centainly for each time
    **/
    for(int nPos = 0; nPos < CHAINLEN; nPos++)
    {
        /**First Step(Cipher Function)**/
        GenerateKey(m_nIndex,roundKeys);
        m_nIndex = DESOneTime(roundKeys);
	m_nIndex &= totalSpace;

        int nnpos = nPos;
        if(nPos < 1300) nnpos = 0;
        m_nIndex = (m_nIndex + nnpos) & totalSpace;
        m_nIndex = (m_nIndex + (nnpos << 8)) & totalSpace;
        m_nIndex = (m_nIndex + ((nnpos << 8) << 8)) & totalSpace;	    
    }

    data[TX] = m_nIndex;

    __syncthreads();
}

__global__ void OneTimeForTotal(uint64_t * in)
{

    for(int i=0; i<256; i++)
    {
        ((uint64_t *)des_SP)[i] = ((uint64_t *)des_d_sp_c)[i];
    }
    uint64_t roundKeys[16], key = 0x020E0E0E0E0E0E0E;
    GenerateKey(key, roundKeys);

    in[TX]=DESOneTime(roundKeys);
}

void KeyTest();

void OneTimeTestForTotal()
{

    KeyTest();

    uint64_t * cudaIn;
    uint64_t starts[1];
    _CUDA(hipMalloc((void**)&cudaIn , sizeof(uint64_t)*1));
    OneTimeForTotal<<<1, 1>>>(cudaIn); cout << "hello" << endl;
    cudaerrno=hipGetLastError();
    if(hipSuccess!=cudaerrno)
    {
        fprintf(stderr, "Cuda error %d in file '%s' in line %i: %s\n",cudaerrno,__FILE__,__LINE__,hipGetErrorString(cudaerrno));
        exit(EXIT_FAILURE);
    }

    _CUDA(hipMemcpy(starts,cudaIn,sizeof(uint64_t)*1,hipMemcpyDeviceToHost));
    cout<<starts[0]<<endl;
}

void OneTimeTest()
{
    uint64_t incuda[16];

    _CUDA(hipMalloc((void**)&incuda , sizeof(uint64_t)*16));
    DESEncrypt<<<1,1>>>(incuda);

    FILE*file=fopen("OneTimeTest.txt","wb");
    assert(file);
    for(int i = 0; i<16; i++)
    {
        fwrite((char*)&(incuda[i]),sizeof(uint64_t),1,file);
    }
    //assert(fwrite((char*)incuda,sizeof(uint64_t),16,file) == 16);
    //fclose(file);

    des_key_schedule ks;//const uint64_t key   = 0xF1F1F1F1F1F1F1F1;
    const_DES_cblock key = {0xF1,0xF1,0xF1,0xF1,0xF1,0xF1,0xF1,0xF1};

    DES_set_key_unchecked(&key,&ks);

    for(int i = 0; i< 16; i++)
    {
        fwrite((char*)&(ks.ks[i].cblock),sizeof(uint64_t),1,file);
    }
}

uint64_t rand64()
{
    uint64_t rs = 0;
    for(uint64_t i = 0; i < 8; i++)
        rs |= ((rand() % 256) << (8*i));
    return rs;
}

#define FF(i, n) for(i = 0;i < n;i++)

/**
	Combined with DESEncrypt to conduct simple performance test
**/
void DESCrypt()
{
    /**struct timeval tstart, tend;**/

    uint64_t * deviceKeyIn, *deviceKeyOut;
    uint64_t   keys[ALL];
    int i;

    int round = 0, size;
    FILE * f1;
    FILE * f2;

    f1 = fopen("start.in" ,"wb");
    f2 = fopen("end.in"   ,"wb");

    assert(f1 && f2);

    printf("Starting DES kernel\n");

    size = ALL * sizeof(uint64_t);

    _CUDA(hipMalloc((void**)&deviceKeyIn , size));
    _CUDA(hipMalloc((void**)&deviceKeyOut, size));

    while(1)
    {
        printf("Begin Round: %d\n",round);

        fprintf(f1,"Begin Round: %d\n",round);
        fprintf(f2,"Begin Round: %d\n",round);

        /**Need further modification to make it os-independent.**/

        /**gettimeofday(&tstart, NULL);**/

        FF(i, ALL) keys[i] = rand64();
        FF(i, ALL) fprintf(f1,"%lld",(long long)keys[i]);

        _CUDA(hipMemcpy(deviceKeyIn, keys, size, hipMemcpyHostToDevice));
        DESEncrypt<<<BLOCK_LENGTH, MAX_THREAD>>>(deviceKeyIn);

        _CUDA(hipMemcpy(keys, deviceKeyOut, size, hipMemcpyDeviceToHost));

        FF(i, ALL) fprintf(f2,"%lld\n", (long long)keys[i]);

        /**gettimeofday(&tend, NULL);

        long long uses=1000000*(tend.tv_sec-tstart.tv_sec)+(tend.tv_usec-tstart.tv_usec);

        printf("round time: %lld us\n", uses);
        fprintf(f1,"round time: %lld us\n",uses);
        fprintf(f2,"round time: %lld us\n",uses);
        **/
        printf("End Round: %d\n",round);
        fprintf(f1,"End Round: %d\n",round);
        fprintf(f2,"End Round: %d\n",round);

        round++;
    }

    //fclose(f1);fclose(f2);

    //printf("Ending DES kernel\n");
}

/**
Combined with DESGeneratorCUDA to generate data
**/

void Logo()
{
    printf("DESRainbowCrack 1.0\n 	Make an implementation of DES Time-and-Memory Tradeoff Technology\n 	By Tian Yulong(mathetian@gmail.com)\n\n");
}

void Usage()
{
    Logo();
    printf("Usage: gencuda   chainLen chainCount suffix\n");
    printf("                 benchmark\n");
    printf("                 onetimetest\n");
    printf("                 keystest\n");
    printf("                 onetimefortotal\n\n");

    printf("example 1: gencuda 1000 10000 suffix\n");
    printf("example 2: gencuda benchmark\n");
}

struct RainbowChain_t
{
    uint64_t nStartKey;
    uint64_t nEndKey;
};

typedef struct RainbowChain_t RainbowChain;

uint64_t GetFileLen(FILE* file)
{
    unsigned int pos = ftell(file);
    fseek(file, 0, SEEK_END);
    uint64_t len = ftell(file);
    fseek(file, pos, SEEK_SET);

    return len;
}
uint64_t Convert(uint64_t num)
{
    uint64_t rs = 0, tmp =0;
    tmp = num & ((1ull << 7) - 1); tmp <<= 1;
    rs = tmp; num >>= 7;
    tmp = num & ((1ull << 7) - 1); tmp <<= 1; tmp <<= 8;
    rs |= tmp; num >>= 7;
    tmp = num & ((1ull << 7) - 1); tmp <<= 1; tmp <<= 16;
    rs |= tmp; num >>= 7;
    tmp = num & ((1ull << 7) - 1); tmp <<= 1; tmp <<= 24;
    rs |= tmp; num >>= 7;
    return rs;
}
void DESGenerator(uint64_t chainLen, uint64_t chainCount, const char * suffix)
{
    char fileName[100];
    memset(fileName, 0, 100);

    sprintf(fileName,"DES_%lld-%lld_%s-cuda", (long long)chainLen, (long long)chainCount,suffix);

    FILE * file = fopen(fileName, "ab+");
    assert(file);

    uint64_t nDatalen = GetFileLen(file);

    assert((nDatalen & ((1 << 4) - 1)) == 0);

    int remainCount =  chainCount - (nDatalen >> 4);

    int time1 = (remainCount + ALL - 1)/ALL;
    /**Start Preparation**/

    uint64_t size = sizeof(uint64_t)*ALL;

    uint64_t * cudaIn;
    uint64_t starts[ALL], ends[ALL];

    _CUDA(hipMalloc((void**)&cudaIn , size));

    /**End Preparation**/
    printf("Need to compute %d rounds %lld\n", time1, (long long)remainCount);

    for(int round = 0; round < time1; round++)
    {
        printf("Begin compute the %d round\n", round+1);

        TimeStamp tms;
        tms.StartTime();

        for(uint64_t i = 0; i < ALL; i++)
        {
            starts[i] = Convert(round*ALL + i);
	    starts[i] &= totalSpaceT;
	}
        /**Belong to CUDA logic**/
        _CUDA(hipMemcpy(cudaIn,starts,size,hipMemcpyHostToDevice));

        DESGeneratorCUDA<<<BLOCK_LENGTH, MAX_THREAD>>>(cudaIn);

        _CUDA(hipMemcpy(ends,cudaIn,size,hipMemcpyDeviceToHost));
        /**End of CUDA logic**/

        for(uint64_t i = 0; i < ALL; i++)
        {
            /**Soooory for the sad expression**/
            int flag1 = fwrite((char*)&(starts[i]),sizeof(uint64_t),1,file);
            int flag2 = fwrite((char*)&(ends[i]),sizeof(uint64_t),1,file);
            assert((flag1 == 1) && (flag2 == 1));
        }

        printf("End compute the %d round\n", round+1);
        tms.StopTime("StopTime: ");
    }
}

void KeyTest()
{
    uint64_t * cudaIn;
    uint64_t starts[16];
    _CUDA(hipMalloc((void**)&cudaIn , sizeof(uint64_t)*16));
    Gee<<<1, 1>>>(cudaIn); cout << "hello" << endl;
    _CUDA(hipMemcpy(starts,cudaIn,sizeof(uint64_t)*16,hipMemcpyDeviceToHost));
    for(int i=0; i<16; i++)
    {
        cout<<starts[i]<<endl;
    }
}

#ifdef _WIN32
inline uint64_t atoll(const char * str)
{
    uint64_t rs;
    istringstream ist(str);
    ist >> rs;

    return rs;
}
#endif

int main(int argc, char * argv[])
{
    if(argc != 2 && argc != 4)
    {
        Usage();
        return 1;
    }

    if(argc == 2)
    {
        if(strcmp(argv[1],"benchmark") == 0)
            DESCrypt();
        else if(strcmp(argv[1],"onetimetest") == 0)
            OneTimeTest();
        else if(strcmp(argv[1],"keystest")==0)
            KeyTest();
        else if(strcmp(argv[1],"onetimefortotal")==0)
            OneTimeTestForTotal();
        else Usage();
        return 1;
    }

    uint64_t chainLen, chainCount;
    char suffix[100];

    memset(suffix,0,sizeof(suffix));

    chainLen   = atoll(argv[1]);
    chainCount = atoll(argv[2]);
    memcpy(suffix,argv[3],strlen(argv[3]));

    DESGenerator(chainLen, chainCount, suffix);

    return 0;
}
